
#include <hip/hip_runtime.h>
#include <cstdio>
#include <cstdlib>     // srand, rand
#include <ctime>       // time
#include <sys/time.h>  // get_wall_time
#include <time.h>

#define IMDEP 256
#define SIZE (100*1024*1024) // 100 MB

#define NBLOCKS 32
#define THREADS_PER_BLOCK 100

const int numRuns = 10;

double get_wall_time(){
    struct timeval time;
    if (gettimeofday(&time,NULL)){
        printf("Error en la medicion de tiempo CPU!!\n");
        return 0;
    }
    return (double)time.tv_sec + (double)time.tv_usec * .000001;
}

void* inicializarImagen(unsigned long nBytes){
        unsigned char * img = (unsigned char*) malloc( nBytes );
        for(unsigned long i = 0; i<nBytes; i++){
                img[i] = rand() % IMDEP;
        }
        return img;     
}

void histogramaCPU(unsigned char* img, unsigned long nBytes, unsigned int* histo){
        for(int i = 0; i<IMDEP; i++)
                histo[i] = 0; //Inicializacion

        double wall0, wall1;

        wall0 = get_wall_time();
        for(unsigned long i = 0; i<nBytes; i++)
                histo[img[i]]++;
        
        wall1 = get_wall_time();

        printf("Tiempo de CPU (s): %.4lf\n", wall1 - wall0);
}

long calcularCheckSum(unsigned int* histo){
        long checkSum = 0;
        for(int i = 0; i<IMDEP; i++){
                checkSum += histo[i];
        }
        return checkSum;
}

int compararHistogramas(unsigned int* histA, unsigned int* histB){
        int valido = 1; 
        for(int i = 0; i<IMDEP; i++){
                if(histA[i] != histB[i]){
                        printf("Error en [%d]: %u != %u\n", i, histA[i], histB[i]);
                        valido = 0;
                }
        }
        return valido;
}

__global__ void kernelHistograma(unsigned char *imagen, unsigned long size, unsigned int* histo){
       
        __shared__ unsigned int temp[IMDEP];
        temp[threadIdx.x] = 0;
        __syncthreads();

        unsigned long i = threadIdx.x + blockIdx.x * blockDim.x;
        int offset = blockDim.x * gridDim.x;

        while (i < size){
                atomicAdd( &temp[imagen[i]], 1);
                i += offset;
        }
        
        __syncthreads();
        
        int div = IMDEP / THREADS_PER_BLOCK,
            remainder = IMDEP % THREADS_PER_BLOCK,
            send_id;
        
        int num_sends = (threadIdx.x < remainder) ? div + 1 : div;
        
        for (int i = 0; i < num_sends; i++){
                send_id = (!remainder) ? i + threadIdx.x*div : 
                          (threadIdx.x < remainder ? i + div*threadIdx.x + threadIdx.x 
                                : i + div*threadIdx.x + remainder);
                
                atomicAdd( &(histo[send_id]), temp[send_id] );
        }
}

int main(void){
        if (THREADS_PER_BLOCK > IMDEP){
                printf("\nNo sense in using BLOCK_SIZE greater than IMDEP (%d)\n\n", IMDEP);
                exit (-1);
        }
        
        unsigned char* imagen = (unsigned char*) inicializarImagen(SIZE);
        unsigned int histoCPU[IMDEP];
        histogramaCPU(imagen, SIZE, histoCPU);
        long chk = calcularCheckSum(histoCPU);
        printf("Check-sum CPU: %ld\n", chk);

        unsigned char *dev_imagen = 0;
        unsigned int *dev_histo = 0;
        hipMalloc( (void**) &dev_imagen, SIZE );
        hipMemcpy( dev_imagen, imagen, SIZE, hipMemcpyHostToDevice );
        hipMalloc( (void**) &dev_histo, IMDEP * sizeof( unsigned int) );

        hipEvent_t start, stop;
        hipEventCreate(&start);
        hipEventCreate(&stop);
        float milliSeconds = 0.0;
        float aveGPUMS = 0.0;

        for(int iter = -1; iter<numRuns; iter++){ //La iteracion -1 es para que la tarjeta se ponga en marcha, normalmente siempre da peores tiempos.
                hipMemset( dev_histo, 0, IMDEP * sizeof( unsigned int ) );
                if(iter<0){
                        kernelHistograma<<<NBLOCKS, THREADS_PER_BLOCK>>>(dev_imagen, SIZE, dev_histo);
                }else{
                        hipDeviceSynchronize();
                        hipEventRecord(start);
                        kernelHistograma<<<NBLOCKS, THREADS_PER_BLOCK>>>(dev_imagen, SIZE, dev_histo);
                        hipEventRecord(stop);
                        hipEventSynchronize(stop);
                        hipEventElapsedTime(&milliSeconds, start, stop);
                        aveGPUMS += milliSeconds;
                }
        }
        
        hipEventDestroy(start);
        hipEventDestroy(stop);

        unsigned int gpuHisto[IMDEP];
        hipMemcpy(gpuHisto, dev_histo, IMDEP * sizeof(unsigned int), hipMemcpyDeviceToHost );
        chk = calcularCheckSum(gpuHisto);
        printf("Check-sum GPU: %ld\n", chk);
        
        if(compararHistogramas(histoCPU, gpuHisto))
                printf("Calculo correcto!!\n");

        printf("Tiempo medio de ejecucion del kernel<<<%d, %d>>> sobre %u bytes [s]: %.4f\n", NBLOCKS, THREADS_PER_BLOCK, SIZE, aveGPUMS / (numRuns*1000.0));

        
        hipFree(dev_imagen);
        hipFree(dev_histo);
        free(imagen);

        return 0;
}
